#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////
#define SCAN_BLOCK_DIM 1024
#define NUM_CIRCLES_PER_BLOCK SCAN_BLOCK_DIM
#define TILE_SIZE 64

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;
    short* circleFlags;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"
#include "circleBoxTest.cu_inl"
#include "exclusiveScan.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
// 
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work 
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position 
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length 
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus 
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() { 
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition 
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball 
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
// __device__ __inline__ void
// shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

//     float diffX = p.x - pixelCenter.x;
//     float diffY = p.y - pixelCenter.y;
//     float pixelDist = diffX * diffX + diffY * diffY; // calculate squared distance between circle center and pixel center.

//     float rad = cuConstRendererParams.radius[circleIndex];;
//     float maxDist = rad * rad;

//     // circle does not contribute to the image
//     if (pixelDist > maxDist) // distance is greater than the circle's radius, and so circle doesn't contribute to the pixel.
//         return;

//     float3 rgb;
//     float alpha;

//     // there is a non-zero contribution.  Now compute the shading value

//     // suggestion: This conditional is in the inner loop.  Although it
//     // will evaluate the same for all threads, there is overhead in
//     // setting up the lane masks etc to implement the conditional.  It
//     // would be wise to perform this logic outside of the loop next in
//     // kernelRenderCircles.  (If feeling good about yourself, you
//     // could use some specialized template magic).
//     if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

//         const float kCircleMaxAlpha = .5f;
//         const float falloffScale = 4.f;

//         float normPixelDist = sqrt(pixelDist) / rad;
//         rgb = lookupColor(normPixelDist);

//         float maxAlpha = .6f + .4f * (1.f-p.z);
//         maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
//         alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

//     } else {
//         // simple: each circle has an assigned color
//         int index3 = 3 * circleIndex;
//         rgb = *(float3*)&(cuConstRendererParams.color[index3]);
//         alpha = .5f;
//     }

//     float oneMinusAlpha = 1.f - alpha;

//     // BEGIN SHOULD-BE-ATOMIC REGION --> THIS IS WHERE THE ATOMICITY ISSUES ARISE
//     // global memory read

//     float4 existingColor = *imagePtr;
//     float4 newColor;
//     newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
//     newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
//     newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
//     newColor.w = alpha + existingColor.w;

//     // global memory write
//     *imagePtr = newColor; 

//     // END SHOULD-BE-ATOMIC REGION
// }

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
// __global__ void kernelRenderCircles() {

//     int index = blockIdx.x * blockDim.x + threadIdx.x; //each thread is reponsible for rendering a circle.

//     if (index >= cuConstRendererParams.numCircles)
//         return;

//     int index3 = 3 * index; 

//     // read position (x, y, and z) and radius
//     float3 p = *(float3*)(&cuConstRendererParams.position[index3]); // float3 holds 3 components, p.x, p.y, and p.z
//     float  rad = cuConstRendererParams.radius[index];

//     // compute the bounding box of the circle. The bound is in integer
//     // screen coordinates, so it's clamped to the edges of the screen.
//     // px, py are the x,y coordinates of the center of the circle.
//     short imageWidth = cuConstRendererParams.imageWidth;
//     short imageHeight = cuConstRendererParams.imageHeight;
//     short minX = static_cast<short>(imageWidth * (p.x - rad)); 
//     short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1; // include partially covered pixels
//     short minY = static_cast<short>(imageHeight * (p.y - rad));
//     short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1; // include partially covered pixels, helps with inclusive loop bounds

//     // a bunch of clamps.  Is there a CUDA built-in for this?
//     short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
//     short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
//     short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
//     short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

//     float invWidth = 1.f / imageWidth;
//     float invHeight = 1.f / imageHeight;

//     // for all pixels in the bonding box --> THIS IS WHERE THE ATOMICITY AND ORDER VIOLATIONS OCCUR
//     for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
//         float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]); // pointer to RGB data for the pixel located at screenMinX, pixelY. We increment to the next pixel in the row
//         for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
//             float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
//                                                  invHeight * (static_cast<float>(pixelY) + 0.5f)); // normalize pixel width and height by image width and height. Turn to 2 component float.
//             shadePixel(index, pixelCenterNorm, p, imgPtr); // call shadePixel for the circle, the normalized bounding box widths, the circle position
//             imgPtr++; // We increment to the next pixel in the row
//         }
//     }
// }

// Applies the changes induced by a batch of circles for a pixel.
__global__ void kernelRenderCircles(int offset, int numTileCircles) {
    int pixelIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int colorOffset = 4 * pixelIndex;
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    int numPixels = imageWidth*imageHeight;
    if (pixelIndex >= numPixels) return;


    // Calculate pixel coordinates
    short pixelY = pixelIndex / imageWidth;
    short pixelX = pixelIndex % imageWidth;

    float invWidth = 1.f/imageWidth;
    float invHeight = 1.f/imageHeight;
    
    float2 pixelCenterNorm = make_float2(
        invWidth * (static_cast<float>(pixelX) + 0.5f),
        invHeight * (static_cast<float>(pixelY) + 0.5f)
    );

    // Determine tile index
    short numTilesX = (imageWidth + TILE_SIZE- 1) / TILE_SIZE;
    short tileIndex = (pixelY / TILE_SIZE) * numTilesX + (pixelX / TILE_SIZE);

    float4 pixelColor = make_float4(0.f, 0.f, 0.f, 0.f);
    pixelColor.x = cuConstRendererParams.imageData[colorOffset];
    pixelColor.y = cuConstRendererParams.imageData[colorOffset + 1];
    pixelColor.z = cuConstRendererParams.imageData[colorOffset + 2];
    pixelColor.w = cuConstRendererParams.imageData[colorOffset+ 3];


    for (int idx = 0; idx < numTileCircles; ++idx) {
        int flagIndex = NUM_CIRCLES_PER_BLOCK * tileIndex + idx;
        short flagValue = cuConstRendererParams.circleFlags[flagIndex];
        if (flagValue < 0) break;

        int circleIndex = offset + flagValue;

        int posIndex3 = 3 * circleIndex;

        // Read position and radius
        float3 p = *(float3*)(&cuConstRendererParams.position[posIndex3]);
        float rad = cuConstRendererParams.radius[circleIndex];
        float maxDist = rad * rad;

        float diffX = p.x - pixelCenterNorm.x;
        float diffY = p.y - pixelCenterNorm.y;
        float pixelDist = diffX*diffX + diffY*diffY;
        
        if (pixelDist > maxDist) continue;

        float3 rgb;
        float alpha;

        if (cuConstRendererParams.sceneName == SNOWFLAKES ||
            cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {
    
            const float kCircleMaxAlpha = 0.5f;
            const float falloffScale = 4.0f;

            // Compute the RGBA of the circle for this pixel
            float rad = cuConstRendererParams.radius[circleIndex];
            float normPixelDist = sqrtf(pixelDist) / rad;
            rgb = lookupColor(normPixelDist);

            float maxAlpha = 0.6f + 0.4f * (1.0f - p.z);
            maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.0f), 0.0f);
            alpha = maxAlpha * expf(-falloffScale * normPixelDist * normPixelDist);
        } else {
            int colorIndex3 = 3 * circleIndex;
            rgb = *(float3*)(&cuConstRendererParams.color[colorIndex3]);
            alpha = .5f;
        }

        float oneMinusAlpha = 1.0f - alpha;
        pixelColor.x = alpha * rgb.x + oneMinusAlpha * pixelColor.x;
        pixelColor.y = alpha * rgb.y + oneMinusAlpha * pixelColor.y;
        pixelColor.z = alpha * rgb.z + oneMinusAlpha * pixelColor.z;
        pixelColor.w = alpha + pixelColor.w;
    }

    cuConstRendererParams.imageData[colorOffset] = pixelColor.x;
    cuConstRendererParams.imageData[colorOffset + 1] = pixelColor.y;
    cuConstRendererParams.imageData[colorOffset+ 2] = pixelColor.z;
    cuConstRendererParams.imageData[colorOffset + 3] = pixelColor.w;

}

__global__ void setTileCircles(int offset, int numTileCircles) {
    short circleIdx = threadIdx.x;
    int circleIndex = offset + circleIdx;
    
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    float normTileMinX = invWidth*static_cast<float>(TILE_SIZE * blockIdx.x);
    float normTileMaxX = invWidth*static_cast<float>(TILE_SIZE*blockIdx.x + TILE_SIZE);
    float normTileMinY = invHeight*static_cast<float>(TILE_SIZE * blockIdx.y);
    float normTileMaxY = invHeight*static_cast<float>(TILE_SIZE*blockIdx.y + TILE_SIZE);

    int posIndex3 = 3 * circleIndex;
    float3 p = *(float3*)(&cuConstRendererParams.position[posIndex3]);
    float rad = cuConstRendererParams.radius[circleIndex];

    short circleInTileFlag = (circleIndex >= cuConstRendererParams.numCircles) ? 0 : circleInBox(p.x, p.y, rad, normTileMinX, normTileMaxX, normTileMaxY, normTileMinY);


    __shared__ uint scanInput[SCAN_BLOCK_DIM];
    scanInput[circleIdx] = circleInTileFlag;

    __syncthreads();  // Wait until all circles for the tile finish

    __shared__ uint scanOutput[SCAN_BLOCK_DIM];
    __shared__ uint scanIntermediates[2 * SCAN_BLOCK_DIM];
    sharedMemExclusiveScan(circleIdx, scanInput, scanOutput, scanIntermediates, SCAN_BLOCK_DIM);

    if(circleIndex >= cuConstRendererParams.numCircles) return; 

    short numTilesX = (imageWidth + TILE_SIZE- 1) / TILE_SIZE;

    short tileIndex = blockIdx.y * numTilesX + blockIdx.x;

    int flagIndex = NUM_CIRCLES_PER_BLOCK * tileIndex + scanOutput[circleIdx];

    if (circleIdx < numTileCircles - 1 && scanOutput[circleIdx + 1] != scanOutput[circleIdx]) {
        cuConstRendererParams.circleFlags[flagIndex] = circleIdx;
    }
    else if (circleIdx == numTileCircles - 1) {
        cuConstRendererParams.circleFlags[flagIndex] = scanInput[circleIdx] ? circleIdx : -1;
        
        if (scanInput[circleIdx] && scanOutput[circleIdx] < circleIdx) {
            cuConstRendererParams.circleFlags[flagIndex + 1] = -1;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);

    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    int numTilesX = (image->width + TILE_SIZE- 1) / TILE_SIZE;
    int numTilesY = (image->height + TILE_SIZE - 1) / TILE_SIZE;

    hipMalloc(&cudaDeviceCircleFlags, sizeof(short)* NUM_CIRCLES_PER_BLOCK * numTilesX * numTilesY);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;
    params.circleFlags = cudaDeviceCircleFlags;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();
}

// void CudaRenderer::render() {

//     // 256 threads per block is a healthy number
//     dim3 blockDim(256, 1);
//     dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

//     kernelRenderCircles<<<gridDim, blockDim>>>();
//     hipDeviceSynchronize();
// }
void CudaRenderer::render() {
  // 256 threads per block is a healthy number
  dim3 blockDim(256, 1);
  dim3 pixelDim((image->width * image->height + blockDim.x - 1) / blockDim.x);

  short numTilesX = (image->width + TILE_SIZE- 1) / TILE_SIZE;
  short numTilesY = (image->height + TILE_SIZE - 1) / TILE_SIZE;
  dim3 tileGridDim(numTilesX, numTilesY);

  for (int i = 0; i < numCircles; i += NUM_CIRCLES_PER_BLOCK) {
    setTileCircles<<<tileGridDim, NUM_CIRCLES_PER_BLOCK>>>(i, std::min(NUM_CIRCLES_PER_BLOCK, numCircles - i));
    kernelRenderCircles<<<pixelDim, blockDim>>>(i, std::min(NUM_CIRCLES_PER_BLOCK, numCircles - i));
    hipDeviceSynchronize();
  }
}